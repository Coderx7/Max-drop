#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/spatial_dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

	template <typename Dtype>
	__global__ void SpatialDropoutForward(const int nthreads,
		const int height, const int width, const Dtype* in,
		const unsigned int* mask, const unsigned int threshold, const float scale,
		Dtype* out){
		CUDA_KERNEL_LOOP(index, nthreads){
			Dtype mulVal = Dtype(0);
			if (mask[index] > threshold){
				mulVal = Dtype(1);
			}
			for (int h = 0; h < height; h++){
				for (int w = 0; w < width; w++){
					int index_in = (index*height + h)*width + w;
					out[index_in] = in[index_in] * mulVal * scale;
				}
			}
		}
	}

	template <typename Dtype>
	void SpatialDropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();
		const int count = bottom[0]->count();
		if (this->phase_ == TRAIN) {
			const int num = bottom[0]->num();
			const int channels = bottom[0]->channels();
			const int width = bottom[0]->width();
			const int height = bottom[0]->height();
			unsigned int* mask =
				static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
			caffe_gpu_rng_uniform(num*channels, mask);
			// set thresholds
			// NOLINT_NEXT_LINE(whitespace/operators)
			SpatialDropoutForward<Dtype> << <CAFFE_GET_BLOCKS(num*channels), CAFFE_CUDA_NUM_THREADS >> >(
				num*channels, height, width, bottom_data, mask, uint_thres_, scale_, top_data);
			CUDA_POST_KERNEL_CHECK;
		}
		else {
			caffe_copy(count, bottom_data, top_data);
		}
	}

	template <typename Dtype>
	__global__ void SpatialDropoutBackward(const int nthreads,
		const int height, const int width, const Dtype* in_diff,
		const unsigned int* mask, const unsigned int threshold, const float scale,
		Dtype* out_diff){
		CUDA_KERNEL_LOOP(index, nthreads) {
			Dtype mulVal = Dtype(0);
			if (mask[index] > threshold){
				mulVal = Dtype(1);
			}
			for (int h = 0; h < height; h++){
				for (int w = 0; w < width; w++){
					int index_in = (index*height + h)*width + w;
					out_diff[index_in] = in_diff[index_in] * scale * mulVal;
				}
			}
		}
	}

	template <typename Dtype>
	void SpatialDropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		if (propagate_down[0]) {
			const Dtype* top_diff = top[0]->gpu_diff();
			Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
			if (this->phase_ == TRAIN) {
				const int num = bottom[0]->num();
				const int channels = bottom[0]->channels();
				const int width = bottom[0]->width();
				const int height = bottom[0]->height();
				const unsigned int* mask =
					static_cast<const unsigned int*>(rand_vec_.gpu_data());
				const int count = bottom[0]->count();
				// NOLINT_NEXT_LINE(whitespace/operators)
				SpatialDropoutBackward<Dtype> << <CAFFE_GET_BLOCKS(num*channels),
					CAFFE_CUDA_NUM_THREADS >> >(
					num*channels, height, width, top_diff, mask, uint_thres_, scale_, bottom_diff);
				CUDA_POST_KERNEL_CHECK;
			}
			else {
				caffe_copy(top[0]->count(), top_diff, bottom_diff);
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(SpatialDropoutLayer);

}  // namespace caffe
